#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <thread>
#include <mutex>
#include "..\..\include\PhotronVideoCapture.h"

#include "hip/hip_runtime.h"
#include "opencv2/core.hpp"
#include "opencv2/objdetect/objdetect.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/cudaobjdetect.hpp"
#include "opencv2/cudaimgproc.hpp"
#include "opencv2/cudawarping.hpp"
#include "opencv2/cudaarithm.hpp"

#define DETECTION_SCALE 1.0
#define MINOBJSIZE 60
// #define FPS30

using namespace std;
using namespace cv;
using namespace cv::cuda;

bool doCameraInit;
bool cpuMode; // true: cpu, false: gpu
bool canUseGPU;

static int numFramesProcessed = 0;
static int currentCPUSeqNumber = -1;
static int currentGPUSeqNumber = -1;

static unsigned int width = 1246;
static unsigned int height = 1008;
cv::CascadeClassifier cascade_cpu;
static Ptr<cuda::CascadeClassifier> cascade_gpu;

static double scaleFactor = 5.0;
static int minNeighbors = 4;
static int minObjSize = MINOBJSIZE;

static std::vector<cv::Rect> faces;
static std::vector<cv::Rect> facesGUI;
static unsigned char* dst_gpu;
static Mat dst_cpu;
static bool capturing;
static bool recReady;
static bool recTriggered;
mutex mtx;

#define MAX_REC_DURATION 256

static std::vector <GpuMat> fBufGPU;
static std::vector <Mat> fBufCPU;
static std::vector < std::vector < cv::Rect > > facesVec;
static std::vector < SYSTEMTIME > stVec;

photron::PUCLib_Wrapper capGPU;
photron::VideoCapture capCPU;
Mat fullFrame;

const int xMargin = 8;
const int yMargin = 8;
const int xDiv = 6;
const int yDiv = 11;
int yTile;
int xTile;
static int recDuration;
int currentCPUGPU; // cpu = 0, gpu = 1, undefined = -1

static bool loopPlay;

unsigned char* dst;

shared_ptr<std::thread> thCapture = shared_ptr<std::thread>();

static void shutdownCaptureInstance()
{
	if(currentCPUGPU == 0)  // closing cpu mode 
	{
		fprintf(stderr, "Closing CPU mode\n");
		if(thCapture.get() != nullptr) {
			fprintf(stderr, "joining CPU capture thread\n");
			thCapture->join();
		}
	} 
	else if(currentCPUGPU == 1) // closing gpu mode
	{ 
		if(capGPU.isOpened()) 
		{
			fprintf(stderr, "Closing GPU mode ....");
			PUC_EndXferData(capGPU.getPUCHandle());
			capGPU.tearDownGPUDecode();
			capGPU.close();
			fprintf(stderr, "GPU is closed\n");
		}
	}
	currentCPUGPU = -1;
}

static void on_mouse(int event, int x, int y, int flags, void *)
{
	if(event == EVENT_LBUTTONDOWN)
	{
		int jj = (y - yMargin) / yTile;
		int ii = (x - xMargin) / xTile;

		if(jj == 10 && ii >= 4) 
		{	// Exit button
			if(!loopPlay) 
			{ 
				// Not memory playback
				capturing = false;
				recReady = false;
				recTriggered = false;
				shutdownCaptureInstance();    
			} 
			else 
			{	// memory playback
				loopPlay = false;
			}
			free(dst);
			exit(0);
		}

		if(jj == 0)
		{
			if(ii < 3)
			{
				if(!cpuMode)
				{
					cpuMode = true;
					loopPlay = false;
					doCameraInit = true;
				}
			}
			else
			{
				if(canUseGPU){
					if(cpuMode){
						cpuMode = false;
						loopPlay = false;
						doCameraInit = true;
					}
				}
			}
		}
		else if(jj==2)
		{
			if(ii <= 1) {
				scaleFactor -= 0.1;
				if(scaleFactor < 1.2) {
					scaleFactor = 1.2;
				}
			}
			else if(ii >= 4)
			{
				scaleFactor += 0.1;
				if(scaleFactor > 10.0) {
					scaleFactor = 10.0;
				}
			}
		}
		else if(jj==4)
		{
			if(ii <= 1) {
				minNeighbors -= 1;
				if(minNeighbors < 0) {
					minNeighbors = 0;
				}
			}
			else if(ii >= 4) {
				minNeighbors += 1;
				if(minNeighbors > 100){
					minNeighbors = 100;	  
				}
			}
		}
		else if(jj==6)
		{
			if(ii <= 1) {
				minObjSize -= 5;
				if(minObjSize < 5.0) {
					minObjSize = 5;
				}
			}
			else if(ii >= 4) {
				minObjSize += 5;
				if(minObjSize > 300.0) {
				  minObjSize = 300;
				}
			}
		}
		else if(jj==8) 
		{
			if(ii <= 1) {
				recDuration -= 16;
				if(recDuration < 16){
					recDuration = 16;
				}
			}
			else if(ii >= 4) {
				recDuration += 16;
				if(recDuration > MAX_REC_DURATION){
					recDuration = MAX_REC_DURATION;
				}
			}
		}
		else if(jj == 10 && ii == 3) {
			Mat helpPNG;
			std::string fname = samples::findFile("helpPNG.PNG");
			helpPNG = imread(fname);
			imshow("Help", helpPNG);
		}
		else if(jj == 10 && ii == 1) { // Save Param
			FILE *fp;
			fp = fopen("facesParam.txt", "w");
			if(fp != NULL) {
				fprintf(fp, "%lf\n%d\n%d\n%d\n", 
				scaleFactor,
				minNeighbors,
				minObjSize,
				recDuration);
				fclose(fp);
			}
			else {
				fprintf(stderr, "File output error!\n");
			}
		}
		else if(jj == 10 && ii == 0) { // Rec/Cam/Stop
			if(capturing) {
				if(recReady) {
					if(recTriggered) {
						// Rec loop
						fprintf(stderr, "Stop Rec and change mode to Playback\n");	    
						capturing = false;
						recReady = false;
						recTriggered = false;
					}
					else {
					// RecReady loop
					fprintf(stderr, "Change mode to Live\n");
					recReady = false;
					}
				}
				else {
					// Live loop
					fprintf(stderr, "Change mode to recReady\n");
					recReady = true;
				}
			}
			else {
				// Play back
				loopPlay = false;
			}
		}
	}
}

static void monoResize(const GpuMat& src, GpuMat& resized, double scale)
{
    Size sz(cvRound(src.cols * scale), cvRound(src.rows * scale));
    if (scale != 1) {
      // cv::cuda::resize(src, resized, sz);
    }
	else {
      resized = src;
    }
}

static int getElapsedMillisec(SYSTEMTIME &st, SYSTEMTIME &previousSt)
{
	int c_msec = st.wSecond * 1000 + st.wMilliseconds;
	int p_msec = previousSt.wSecond * 1000 + previousSt.wMilliseconds;
	if(c_msec > p_msec) {
		return c_msec - p_msec;
	}
	return 60000 - p_msec + c_msec;
}

// Callback function for CPU
static void capThreadCPU()
{
	while (1)
	{
		if (!capturing)
			return;

		capCPU.read(dst_cpu);
		if (dst_cpu.empty()) {
			cerr << "ERROR! Unable to capture\n";
			return;
		}

		SYSTEMTIME st;
		GetSystemTime(&st);

		Mat resized_cpu;
		dst_cpu.copyTo(resized_cpu);

		USHORT tmpSeqNum = capCPU.getPUCLibWrapper()->getFullSequenceNumber();
		if(tmpSeqNum == currentCPUSeqNumber){
			continue;
		}

		// cascade_cpu.detectMultiScale(resized_cpu, faces, 5.0, 0, CASCADE_SCALE_IMAGE, cv::Size(MINOBJSIZE,MINOBJSIZE));
		cascade_cpu.detectMultiScale(resized_cpu, faces, scaleFactor, minNeighbors, CASCADE_SCALE_IMAGE, cv::Size(minObjSize, minObjSize));
    
		{
			lock_guard<mutex> lock(mtx);
			dst_cpu.copyTo(fullFrame); // todo
			facesGUI = faces;
		}

		if(recReady) 
		{
			if(faces.size() > 0 || recTriggered) 
			{
				recTriggered = true;
				if(fBufCPU.size() < recDuration) {
					Mat tmp;
					resized_cpu.copyTo(tmp);
					fBufCPU.push_back(tmp);
					facesVec.push_back(faces);
					SYSTEMTIME st;
					GetSystemTime(&st);
					stVec.push_back(st);
				}
				else {
					capturing = false;
					recReady = false;
					recTriggered = false;
				}
			}
		}

		numFramesProcessed++;
		currentCPUSeqNumber = tmpSeqNum;
	}
}

// Callback function for GPU
static void capThreadGPU(PPUC_XFER_DATA_INFO info, void* userData)
{	
	if(!capturing)
		return;

	photron::PUCLib_Wrapper *pCap = (photron::PUCLib_Wrapper *) userData;
	auto result = pCap->decodeGPU(false, info->pData, &dst_gpu, width);
	GpuMat frame_gpu(height, width, CV_8UC1, dst_gpu);
	GpuMat resized_gpu;
	monoResize(frame_gpu, resized_gpu, DETECTION_SCALE);

	USHORT tmpSeqNum = info->nSequenceNo;
	if(tmpSeqNum == currentGPUSeqNumber) {
		return;
	}

	cascade_gpu->setFindLargestObject(false);
	cascade_gpu->setScaleFactor(scaleFactor);
	cascade_gpu->setMinNeighbors(minNeighbors); // 4 -> 8 -> 2 -> 4
	cascade_gpu->setMinObjectSize(cv::Size(minObjSize,minObjSize));

	GpuMat faceBuf_gpu;
	cascade_gpu->detectMultiScale(resized_gpu, faceBuf_gpu);
	cascade_gpu->convert(faceBuf_gpu, faces);

	{
		lock_guard<mutex> lock(mtx);
		facesGUI = faces;
	}

	if(recReady)
	{
		if(faces.size() > 0 || recTriggered) {
			recTriggered = true;
			if(fBufGPU.size() < recDuration){
				GpuMat tmp;
				resized_gpu.copyTo(tmp);
				fBufGPU.push_back(tmp);
				facesVec.push_back(faces);
				SYSTEMTIME st;
				GetSystemTime(&st);
				stVec.push_back(st);
			}
			else {
				capturing = false;
				recReady = false;
				recTriggered = false;
			}
		}
	}
  
	numFramesProcessed++;
	currentGPUSeqNumber = tmpSeqNum;
}

static void pendingMenu(cv::Mat &menuBuf, const char *str)
{
	menuBuf.setTo(cv::Scalar(160,160,160));
  
	putText(menuBuf, str, 
		cv::Point(menuBuf.cols / 2 - 50, menuBuf.rows / 2 - 10), 
		FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	cv::imshow("Faces Menu", menuBuf);
	cv::waitKey(1);
}

static void refreshMenu(cv::Mat &menuBuf, bool cpuMode)
{
	menuBuf.setTo(cv::Scalar(100,100,100));

	cv::Point pt0, pt1;
  
	yTile = (menuBuf.rows - yMargin * 2) / yDiv;
	xTile = (menuBuf.cols - xMargin * 2) / xDiv;

	pt0.y = yMargin + 0 * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + 0 * xTile;
	pt1.x = pt0.x + xTile * (xDiv / 2) - 2;

	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, cpuMode ? "[v] CPU" : "[ ] CPU", 
		cv::Point(pt0.x + 25, pt0.y + 20),
		FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);
  
	pt0.y = yMargin + 0 * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + (xDiv / 2) * xTile;
	pt1.x = pt0.x + xTile * (xDiv / 2) - 2;

	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, cpuMode ? "[ ] GPU" : "[v] GPU", 
		cv::Point(pt0.x + 25, pt0.y + 20), 
		FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	int ySection; // 0,1,2,3,4,5
	int xSection; // 0,1,2,3,4,5
	int xWidth;	  // 1,2,3,4,5,6
	char numLabel[8];

	sprintf(numLabel, "%.1f", scaleFactor);

	ySection = 1;
	xSection = 0;
	pt0.y = yMargin + ySection * yTile;
	pt0.x = xMargin + xSection * xTile;
	putText(menuBuf, "Scale Factor" , cv::Point(pt0.x + 5, pt0.y + 30), FONT_HERSHEY_PLAIN, 1.0, Scalar(255,255,255), 1, 8);

	ySection = 3;
	xSection = 0;
	pt0.y = yMargin + ySection * yTile;
	pt0.x = xMargin + xSection * xTile;
	putText(menuBuf, "Min. Neighbor" , cv::Point(pt0.x + 5, pt0.y + 30), FONT_HERSHEY_PLAIN, 1.0, Scalar(255,255,255), 1, 8);

	ySection = 5;
	xSection = 0;
	pt0.y = yMargin + ySection * yTile;
	pt0.x = xMargin + xSection * xTile;
	putText(menuBuf, "Min. Size" , cv::Point(pt0.x + 5, pt0.y + 30), FONT_HERSHEY_PLAIN, 1.0, Scalar(255,255,255), 1, 8);  

	ySection = 7;
	xSection = 0;
	pt0.y = yMargin + ySection * yTile;
	pt0.x = xMargin + xSection * xTile;
	putText(menuBuf, "Rec. Duration" , cv::Point(pt0.x + 5, pt0.y + 30), FONT_HERSHEY_PLAIN, 1.0, Scalar(255,255,255), 1, 8);  

	ySection = 2;
	xSection = 0;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, "<" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 2;
	xSection = 2;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(200,200,200), -1, 4);
	putText(menuBuf, numLabel , cv::Point(pt0.x + 25, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 2;
	xSection = 4;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, ">" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	sprintf(numLabel, "%d", minNeighbors);
	ySection = 4;
	xSection = 0;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, "<" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 4;
	xSection = 2;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(200,200,200), -1, 4);
	putText(menuBuf, numLabel , cv::Point(pt0.x + 25, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 4;
	xSection = 4;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, ">" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	sprintf(numLabel, "%d", minObjSize);
	ySection = 6;
	xSection = 0;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, "<" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 6;
	xSection = 2;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(200,200,200), -1, 4);
	putText(menuBuf, numLabel , cv::Point(pt0.x + 25, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 6;
	xSection = 4;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, ">" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	sprintf(numLabel, "%d", recDuration);
	ySection = 8;
	xSection = 0;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, "<" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);
	ySection = 8;
	xSection = 2;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(200,200,200), -1, 4);
	putText(menuBuf, numLabel , cv::Point(pt0.x + 25, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 8;
	xSection = 4;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, ">" , cv::Point(pt0.x + 30, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	ySection = 10;
	xSection = 3;
	xWidth = 1;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	putText(menuBuf, "(?)" , cv::Point(pt0.x + 10, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(255, 255, 255), 1, 8);

	// Function Button
	ySection = 10;
	xSection = 0;
	xWidth = 1;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;

	if(capturing) {
		if(recReady) {
			if(recTriggered) {
				cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160, 160, 160), -1, 4);
				putText(menuBuf, "STOP" , cv::Point(pt0.x + 2, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(255, 255, 255), 1, 8);
			}
			else {
				cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160, 160, 160), -1, 4);
				putText(menuBuf, "CAM" , cv::Point(pt0.x + 2, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(255, 255, 255), 1, 8);
			}
		}
		else {
			cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(0, 0, 200), -1, 4);
			putText(menuBuf, "REC" , cv::Point(pt0.x + 2, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(255, 255, 255), 1, 8);
		}
	}
	else {
		cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160, 160, 160), -1, 4);
		putText(menuBuf, "CAM" , cv::Point(pt0.x + 2, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(255, 255, 255), 1, 8);
	}

	// Parameter Save
	ySection = 10;
	xSection = 1;
	xWidth = 1;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, "Sav" , cv::Point(pt0.x + 2, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);
  
	// Exit Button
	ySection = 10;
	xSection = 4;
	xWidth = 2;
	pt0.y = yMargin + ySection * yTile;
	pt1.y = pt0.y + yTile - 2;
	pt0.x = xMargin + xSection * xTile;
	pt1.x = pt0.x + xTile * xWidth - 2;
	cv::rectangle(menuBuf, pt0, pt1, cv::Scalar(160,160,160), -1, 4);
	putText(menuBuf, "Exit" , cv::Point(pt0.x + 15, pt0.y + 20), FONT_HERSHEY_PLAIN, 1.0, Scalar(0, 0, 0), 1, 8);

	cv::imshow("Faces Menu", menuBuf);
}


int main()
{
	std::cout << "Start...\n";

	// Check canUseGPU
	int cudaDevCount = 0;
	if ((cudaDevCount = getCudaEnabledDeviceCount()) < 1) {
		cerr << "No GPU found or the library is compiled without CUDA support" << endl;
		canUseGPU = false;
	}
	else {
		cerr << cudaDevCount << " CUDA device found." << endl;
		cv::cuda::printShortCudaDeviceInfo(cv::cuda::getDevice());
		fprintf(stderr, "CAN use GPU\n");
		canUseGPU = true;
	}
  
	// 
	// Initializing haar cascade classifier
	// 
	string cascadeName = "haarcascade_frontalface_default.xml";
	
	// Cascade construction for CPU
	if(!cascade_cpu.load(cascadeName)) {
		cerr << "cpu cascade load error." << endl;
		return -1;
	}

	// Cascade construction for GPU
	if(canUseGPU) {
		cascade_gpu = cuda::CascadeClassifier::create(cascadeName);
		std::cout << "Loaded(CPU/GPU) " << cascadeName << std::endl;
	}

	// 
	// Load default param
	// 
	FILE *fp;
	fp = fopen("facesParam.txt", "r");
	if(fp != NULL) {
		fscanf(fp, "%lf", &scaleFactor);
		fscanf(fp, "%d", &minNeighbors);
		fscanf(fp, "%d", &minObjSize);
		fscanf(fp, "%d", &recDuration);		
		fclose(fp);
		fprintf(stderr, "Paramters are Loaded (%lf, %d, %d, %d)\n",
			scaleFactor,
			minNeighbors,
			minObjSize,
			recDuration);
	}
	else {
		fprintf(stderr, "File input error!\n");
	}

	// 
	// Common camera initialization
	// 
	PUCRESULT result;
  
	// Menu OpenCV window
	cv::namedWindow("Faces Menu", cv::WINDOW_AUTOSIZE | cv::WINDOW_FREERATIO);
	setMouseCallback("Faces Menu", on_mouse);
	cv::Mat menuBuf;
	menuBuf.create(402, 256, CV_8UC3);
	menuBuf.setTo(cv::Scalar(100,100,100));
  
	std::vector <cv::Rect> menuRect;
	cv::Rect tmpRect;
  
	refreshMenu(menuBuf, cpuMode);

	dst = (unsigned char* )malloc(width * height);

	cpuMode = false;
	recDuration = 64;
	currentCPUGPU = -1; // cpu = 0, gpu = 1, undefined = -1
	float procFPS;
  
	while(1)
	{
		doCameraInit = true;
		recReady = false;
		recTriggered = false;
    
		SYSTEMTIME previousSt;
		int previousNumFramesProcessed = 0;
		int FPS_for_calc_FPS = 0;

		while(1)
		{
			// 
			// Restarting Camera 
			// 
			if(doCameraInit) {
				fprintf(stderr, "Chaging CPU/GPU mode\n");
				// Shutting down the previous mode
				capturing = false;
				pendingMenu(menuBuf, "Initializing...");
				shutdownCaptureInstance();    
				capturing = true;

				// Initializing new mode
				if(cpuMode || !canUseGPU) { // CPUmode
					fprintf(stderr, "Initializing CPU mode\n");
					int deviceID = 0;             // 0 = open default camera
					int apiID = cv::CAP_ANY;      // 0 = autodetect default API
					if(!capCPU.isOpened()) {
						capCPU.open(deviceID, apiID);
						if (!capCPU.isOpened()) {
							cerr << "ERROR! Unable to open camera\n";
							menuBuf.setTo(cv::Scalar(0,0,200));
							putText(menuBuf, "Camera Open Error" , cv::Point(10, 40), FONT_HERSHEY_PLAIN, 1.3, Scalar(255,255,255), 1, 8);
							putText(menuBuf, "Hit ESC to exit" , cv::Point(10, 80), FONT_HERSHEY_PLAIN, 1.3, Scalar(255,255,255), 1, 8);
							cv::imshow("Faces Menu", menuBuf);
							cv::waitKey(0);
							return -1;
						}
					}

					capCPU.getPUCLibWrapper()->pause();
#ifdef FPS30	  
					capCPU.getPUCLibWrapper()->setFramerateShutter((UINT32)30, (UINT32)30); // 1000, 2000
#else
					capCPU.getPUCLibWrapper()->setFramerateShutter((UINT32)1000, (UINT32)2000); // 1000, 2000
#endif
					capCPU.getPUCLibWrapper()->resume();
	  
					thCapture = std::make_shared<std::thread>(&capThreadCPU);
					fprintf(stderr, "Starting CPU mode\n");
					cpuMode = true;
					currentCPUGPU = 0;
				}
				else { // GPUmode
					fprintf(stderr, "Initializing GPU mode.");
					// Saving one image only needs single thread mode API enabled
					capGPU.setMultiThread(false);
					// Set Capture Settings
					capGPU.setResolution(width, height);
#ifdef FPS30	  
					capGPU.setFramerateShutter(30, 30);
#else
					capGPU.setFramerateShutter(1000, 2000);
#endif	  
					fprintf(stderr, "..Opening cap instance.");

					// Open Camera
					result = capGPU.open(0);
					if (result != PUC_SUCCEEDED || !capGPU.isOpened()) {
						std::cerr << capGPU.getLastErrorName();
						currentCPUGPU = -1;
						canUseGPU = false;
						fprintf(stderr, "can NOT use GPU (capOpen Error). Trying CPU mode\n");
						cpuMode = true;
						continue;
					}
					fprintf(stderr, ".. Done\n");
					fprintf(stderr, "..Initialize GPU cap.\n");
	  
					// Initialize GPU capture
					result = capGPU.setupGPUDecode(PUC_GPU_SETUP_PARAM{ width, height });
					if (result != PUC_SUCCEEDED) {
						std::cerr << capGPU.getLastErrorName();
						canUseGPU = false;
						fprintf(stderr, "can NOT use GPU (PUC_SetupGPU Error). Trying CPU mode\n");
						currentCPUGPU = -1;
						cpuMode = true;
						continue;
					}
	  
					fprintf(stderr, "..Begin Xfer\n");
					result = PUC_BeginXferData(capGPU.getPUCHandle(), capThreadGPU, (void *) &capGPU);
					if (result != PUC_SUCCEEDED) {
						fprintf(stderr, "PUC_BeginXferData failed! %d. Trying CPU mode", result);
						currentCPUGPU = -1;
						cpuMode = true;
						continue;
					}
					fprintf(stderr, "Device Sync\n");
					hipDeviceSynchronize();
					fprintf(stderr, "Starting GPU mode.\n");
					currentCPUGPU = 1;
				}
				doCameraInit = false;
			}
			// 
			// Camera Restarted
			// 

			// 
			// Acquire the latest frame
			// 
			if(!cpuMode) {
				auto cudaStatus = hipMemcpy(dst, dst_gpu, (size_t)width * height, hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					Sleep(100);
					continue;
				}
				Mat tmp = Mat(height, width, CV_8UC1, dst);
				tmp.copyTo(fullFrame); // TODO
			}

			if(fullFrame.empty())
				continue;
			refreshMenu(menuBuf, cpuMode);

			//
			// FPS measurement
			// 
			char str[64];
			if(FPS_for_calc_FPS % 10 == 0) { // one for every 10 steps
				SYSTEMTIME st;
				GetSystemTime(&st);
				int millisecondElapse = getElapsedMillisec(st, previousSt);
				procFPS = (numFramesProcessed - previousNumFramesProcessed) / (float)millisecondElapse * 1000;
				previousSt = st;
				previousNumFramesProcessed = numFramesProcessed;

				if(numFramesProcessed > 360000000) { // 10 hrs 1000fps
					numFramesProcessed = 0;
					previousNumFramesProcessed = 0;
				}
			}
			if(recReady) {
				if(recTriggered) {
					sprintf(str, "Recording %.2f FPS", procFPS);
					rectangle(fullFrame, cv::Point(0,0), cv::Point(fullFrame.cols-1, 40), cv::Scalar(255), -1);
					putText(fullFrame, str, cv::Point(20, 30), FONT_HERSHEY_PLAIN, 1.5, Scalar(0), 2, 8);
				}
				else {
					rectangle(fullFrame, cv::Point(0,0), cv::Point(fullFrame.cols-1, 80), cv::Scalar(160), -1);
					sprintf(str, "RecReady %.2f FPS", procFPS);
					putText(fullFrame, str, cv::Point(20, 30), FONT_HERSHEY_PLAIN, 1.5, Scalar(255), 2, 8);
				}
			}
			else {
				sprintf(str, "Camera Live %.2f FPS", procFPS);
				putText(fullFrame, str, cv::Point(20, 30), FONT_HERSHEY_PLAIN, 1.5, Scalar(255), 2, 8);
			}

			FPS_for_calc_FPS++;

			// 
			// Drawing Facial Rectangle
			// 
			for(int ii = 0; ii < facesGUI.size(); ii++) {
				cv::Point pt0, pt1;
				pt0.x = facesGUI[ii].x;
				pt0.y = facesGUI[ii].y;
				pt1.x = facesGUI[ii].x+facesGUI[ii].width;
				pt1.y = facesGUI[ii].y+facesGUI[ii].height;
				cv::rectangle(fullFrame, pt0, pt1, cv::Scalar(255), 2);
			}
      
			//
			// Bar chart for num of faces
			// 
			const int sideMargin = 5;
			const int iconRadius = 15;
			const int gap = 2;
			const float xEyePos = 0.4f; // From center . prop Radius
			const float yEyePos = -0.2f; // From center . prop Radius
			const float eyeHeight = 0.4f; // prop Radius
			const float mouthPos = 0.5f;
			const float mouthUp = 0.1f;
			const float mouthWidth = 0.3f;

			for(int ii = 0; ii < facesGUI.size() && ii < 20; ii++) {
				int cx = sideMargin + iconRadius;
				int cy = fullFrame.rows - (sideMargin + iconRadius + (iconRadius * 2 + gap) * ii) - 1;
				cv::circle(fullFrame, cv::Point(cx, cy), iconRadius, cv::Scalar(255), 2);
				int x0 = (int)(iconRadius * xEyePos);
				int yc = (int)(iconRadius * yEyePos + cy);
				int y0 = (int)(iconRadius * eyeHeight / 2);
				cv::line(fullFrame, cv::Point(cx-x0, yc-y0), cv::Point(cx-x0, yc+y0), cv::Scalar(255), 2);
				cv::line(fullFrame, cv::Point(cx+x0, yc-y0), cv::Point(cx+x0, yc+y0), cv::Scalar(255), 2);
				int ycc = (int)(iconRadius * mouthPos + cy);
				int y1 = (int)(iconRadius * mouthUp);
				int x1 = (int)(iconRadius * mouthWidth);
				cv::line(fullFrame, cv::Point(cx-x1, ycc-y1), cv::Point(cx, ycc), cv::Scalar(255), 2);
				cv::line(fullFrame, cv::Point(cx+x1, ycc-y1), cv::Point(cx, ycc), cv::Scalar(255), 2);
			}

			// 
			// Monitor Display
			// 
			imshow("Infinicam: Faces", fullFrame);

			// 
			// Key operation
			// 
			int key = waitKey(33);
			if (key == 27 || !capturing) {
				pendingMenu(menuBuf, "Closing Camera");

				if(!loopPlay) { // Not memory playback
					capturing = false;
					recReady = false;
					recTriggered = false;
					shutdownCaptureInstance();    
				}
				else { // memory playback
					loopPlay = false;
				}

				if(key == 27) {
					free(dst);
					exit(0);
				}
				break;
			}
				
			if(key=='s') { // start recReady
				recReady = true;
			}
		}
    
		// Shutting down the previous mode
		shutdownCaptureInstance();    

		std::cerr << "Starting video playback." << std::endl;
    
		loopPlay = true;
		bool stepFwd = false;
    
		refreshMenu(menuBuf, cpuMode);
    
		// Demo Video Loop 
		int recordCount = cpuMode ? (int)(fBufCPU.size()) : (int)(fBufGPU.size());
		if(recordCount >= 1) {
			do {
				for(int i = 0; i < recordCount; i++) {
					cv::Mat fBufMat;
					if(cpuMode) {
						fBufCPU[i].copyTo(fBufMat);
					}
					else {
						fBufGPU[i].download(fBufMat);
					}

					int sumWidHgt = 0;
					for(int ii = 0; ii < facesVec[i].size(); ii++) {
						cv::Point pt0, pt1;
						pt0.x = facesVec[i][ii].x;
						pt0.y = facesVec[i][ii].y;
						pt1.x = facesVec[i][ii].x+facesVec[i][ii].width;
						pt1.y = facesVec[i][ii].y+facesVec[i][ii].height;
						cv::rectangle(fBufMat, pt0, pt1, cv::Scalar(255), 2);
						sumWidHgt += facesVec[i][ii].width;
						sumWidHgt += facesVec[i][ii].height;
					}

					int elMsec = 0;
					if(i != 0) {
						elMsec = getElapsedMillisec(stVec[i], stVec[0]);
					}

					cv::Point pa, pb;
					pa.x = 0;
					pa.y = 0;
					pb.x = fBufMat.cols * i / recDuration - 1;
					pb.y = 40;
					rectangle(fBufMat, pa, pb, cv::Scalar(160,160,160), -1, 4);

					char str[128];
					if(facesVec[i].size() == 0) {
						sprintf(str, "No face found. Average Size = --- @ %d [msec] %d / %d", elMsec, i + 1, recordCount);
					}
					else if(facesVec[i].size() == 1) {
						sprintf(str, "%d face found. Average Size = %d @ %d [msec] %d / %d", (int)facesVec[i].size(), sumWidHgt/2, elMsec, i + 1, recordCount);
					}
					else {
						auto aveSize = sumWidHgt / facesVec[i].size() / 2;
						sprintf(str, "%d faces found. Average Size = %d @ %d [msec] %d / %d", (int)facesVec[i].size(), (int)aveSize, elMsec, i + 1, recordCount);
					}
					putText(fBufMat, str, cv::Point(20, 30), FONT_HERSHEY_PLAIN, 1.5, Scalar(255), 2, 8);
					imshow("Infinicam: Faces", fBufMat);
					
					int key = waitKey(stepFwd ? 0 : 33);
					if (key == 27) {
						loopPlay = false;
					}
					if (key == ' ') {
						stepFwd = !stepFwd;
					}
					if (!loopPlay) {
						break;
					}
					refreshMenu(menuBuf, cpuMode);
				}
			} while(loopPlay);

			fprintf(stderr, "Exit from loopPlay\n");

			// Cleanup loop play buffer
			facesVec.clear();
			if(cpuMode)
				fBufCPU.clear();
			else 
				fBufGPU.clear();
		}
	}
}

#if 0

 Faces Help
 
 ---------------------------------------------------------------------------------------------
 * [ ]CPU / [ ]GPU
 CPUプロセスとGPUプロセスを切り替えます。映像デコードと顔検出処理の両方が切り替わります。
 ---------------------------------------------------------------------------------------------
 * Scale Factor
 顔検出の際、基準の大きさにこの因子を繰り返し掛け合わせたサイズで探索を行います。通常1.1などの因子を用いて
 徐々にサイズを変えて探索するほうが検出精度が上がりますが、処理速度が極端に遅くなります。逆に5などの大きい
 値を用いると探索繰り返し回数が少なくなります。検知対象の大きさがある程度限定できる場合には後述の
 Min. Sizeパラメータを調節したうえでScale Factorは大きい値にすると処理が高速化します。
 ---------------------------------------------------------------------------------------------
 * Min. Neighbor
 顔のある場所で、探索窓の位置が少しだけずれても、窓の枠内に目や鼻などの要素が入っていればも顔と認識します。
 すなわち一つの顔の近辺で、少しずつずれた位置に多数検出を行います。Min. Neighborは近傍にいくつの顔が検出
 されれば本当に顔であるかどうかを決めるパラメータです。このパラメータをゼロにすると、顔ではないものを顔と
 認識する誤検出が発生しやすくなります。また、一つの顔の周りに少しずつずれた窓を多数検出します。一方、この
 パラメータを大きくすると検出もれが発生しやすくなります。
 ---------------------------------------------------------------------------------------------
 * Min. Size
 検出する顔の窓の大きさの最小サイズをピクセル単位で設定します。このサイズを小さくすると画面内で小さいサイ
 ズの顔も検出できるようになりますが、処理速度が低下します。検知対象の大きさがある程度限定できる場合には大き
 いサイズにすると高速化します。
 ---------------------------------------------------------------------------------------------
 * Rec. Duration
 RECボタンをクリックするとRecReady状態になります。顔が一つでも検出されるとそれがトリガーとなって録画を
 開始します。Rec. Durationは録画の長さ(フレーム数)です。
 ---------------------------------------------------------------------------------------------
 * REC/CAM/STOP
 Menuウインドウ左下のボタンは、モードによってREC/CAM/STOPの三種類のボタンに切り替わります。
 
 REC: Liveモード(ソフト起動後のモード)でこのボタンを押すとRecReadyモードになります。顔が一つでも検知さ
 れるとトリガーとなって録画を開始します。
 
 CAM: RecReadyモードまたはPlayBackモードでこのボタンを押すとLiveモードに戻ります。ただしトリガーがかかっ
 て録画が始まっている状態ではSTOPボタンになっています。
 
 STOP: 録画モードでこのボタンを押すとPlayBackモードになります。なお、トリガーがかかって録画が始まると顔が
 検出できなくても録画を継続し、Rec. Durationで決められたフレーム数分の録画が終わると自動的にPlayBackモー
 ドになります。
 
 (PlayBackモードでスペースキーを押すとコマ送り再生となり矢印キーなどで一コマずつ動画を進めることができ
 ます。逆再生はできません。もう一度スペースキーを押すと通常再生に戻ります。)
 ---------------------------------------------------------------------------------------------
 * Sav
 設定パラメータを保存します。次回のアプリケーション起動時にこの保存パラメータを自動読み込みします。
 ---------------------------------------------------------------------------------------------
 * Exit
 アプリケーションを終了します。


#endif
